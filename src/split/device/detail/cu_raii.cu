#include "split/device/detail/cu_raii.cuh"

SPLIT_DEVICE_NAMESPACE_BEGIN

namespace detail
{

namespace cu_raii
{

Stream::Stream()
{
  status = hipStreamCreate(&handle);
}

Stream::~Stream()
{
  join();
  hipStreamDestroy(handle);
}

Stream::operator hipStream_t() const noexcept
{
  return handle;
}

void Stream::join() noexcept
{
  status = hipStreamSynchronize(handle);
}

namespace blas
{
Handle::Handle()
{
  status = hipblasCreate(&handle);
}

Handle::~Handle()
{
  hipblasDestroy(handle);
}

Handle::operator hipblasHandle_t() const noexcept
{
  return handle;
}
}

namespace solver
{
SolverDn::SolverDn()
{
  status = hipsolverDnCreate(&handle);
}

SolverDn::~SolverDn()
{
  hipsolverDnDestroy(handle);
}

SolverDn::operator hipsolverHandle_t() const noexcept
{
  return handle;
}

SolverSp::SolverSp()
{
  status = hipsolverSpCreate(&handle);
}

SolverSp::~SolverSp()
{
  hipsolverSpDestroy(handle);
}

SolverSp::operator hipsolverSpHandle_t() const noexcept
{
  return handle;
}
}  // namespace solver

namespace sparse
{
Handle::Handle()
{
  status = hipsparseCreate(&handle);
}

Handle::~Handle()
{
  hipsparseDestroy(handle);
}

Handle::operator hipsparseHandle_t() const noexcept
{
  return handle;
}

MatrixDescription::MatrixDescription()
{
    hipsparseCreateMatDescr(&description);
}

MatrixDescription::MatrixDescription(hipsparseStatus_t* io_status)
{
    *io_status = hipsparseCreateMatDescr(&description);
}

MatrixDescription::~MatrixDescription()
{
    hipsparseDestroyMatDescr(description);
}

MatrixDescription::operator hipsparseMatDescr_t() const noexcept
{
    return description;
}
}  // namespace sparse
}
}

SPLIT_DEVICE_NAMESPACE_END
