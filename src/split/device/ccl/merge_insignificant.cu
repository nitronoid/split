#include "hip/hip_runtime.h"
#include "split/device/ccl/merge_insignificant.cuh"
#include <cusp/graph/connected_components.h>

SPLIT_DEVICE_NAMESPACE_BEGIN

namespace ccl
{
namespace
{
struct AddPair
{
  using pair = thrust::tuple<real, real>;

  __host__ __device__ pair operator()(const pair& lhs, const pair& rhs) const
  {
    const real x = lhs.get<0>() + rhs.get<0>();
    const real y = lhs.get<1>() + rhs.get<1>();
    // Add both channels
    return thrust::make_tuple(x, y);
  }
};

struct ChominanceDistance2
{
  __host__ __device__ thrust::tuple<real, int>
  operator()(const thrust::tuple<thrust::tuple<real, real>,
                                 thrust::tuple<real, real>,
                                 int>& tup) const
  {
    // distance in x
    const real x = tup.get<0>().get<0>() - tup.get<1>().get<0>();
    // distance in y
    const real y = tup.get<0>().get<1>() - tup.get<1>().get<1>();
    // return squared distance and the id
    return thrust::make_tuple(x * x + y * y, tup.get<2>());
  }
};

struct MergeSegment
{
  __host__ __device__ int operator()(const thrust::tuple<int, int>& map,
                                     int target_target) const
  {
    const int current = map.get<0>();
    const int target = map.get<1>();
    // if the target is attempting to merge into us simultaneously, we select
    // the segment with the larger index to prevent oscillations
    if (current == target_target)
      return max(current, target);
    return target;
  }
};

struct TargetMap
{
  const int thresh;

  // Packed => { target, current, size }
  __host__ __device__ int operator()(const thrust::tuple<int, int, int>& tup)
  {
    return tup.get<2>() > thresh ? tup.get<1>() : tup.get<0>();
  }
};

template <typename... Args>
auto zip_it(Args&&... args) -> decltype(
  thrust::make_zip_iterator(thrust::make_tuple(std::forward<Args>(args)...)))
{
  return thrust::make_zip_iterator(
    thrust::make_tuple(std::forward<Args>(args)...));
}

}  // namespace

SPLIT_API void merge_insignificant(
  cusp::array1d<real, cusp::device_memory>::view di_chrominance,
  cusp::array1d<int, cusp::device_memory>::view dio_segment_labels,
  cusp::array1d<int, cusp::device_memory>::view dio_segment_adjacency_keys,
  cusp::array1d<int, cusp::device_memory>::view dio_segment_adjacency,
  cusp::array1d<int, cusp::device_memory>::view dio_segment_size,
  int P)
{
  // Get these sizes upfront
  const int nsegments = dio_segment_adjacency_keys.back() + 1;
  const int npoints = dio_segment_labels.size();
  // Push these into temp storage param eventually
  cusp::array1d<real, cusp::device_memory> total_chrominance(nsegments * 2);
  cusp::array1d<int, cusp::device_memory> indices(npoints);
  cusp::array1d<int, cusp::device_memory> labels(npoints);
  // Target array contains the target segment to join with, initially no change
  cusp::array1d<int, cusp::device_memory> d_targets = dio_segment_labels;

  // Useful iterators

  // Initialize the indices with a standard sequence
  thrust::sequence(indices.begin(), indices.end());
  // Copy our labels for sorting
  thrust::copy(
    dio_segment_labels.begin(), dio_segment_labels.end(), labels.begin());
  // Sort the indices using the labels
  thrust::sort_by_key(labels.begin(), labels.begin(), indices.begin());

  // Counting iterator
  auto count = thrust::make_counting_iterator(0);
  // Segment size range iterators
  auto size_begin = dio_segment_size.begin();
  auto size_end = dio_segment_size.end();
  // Compute the segment sizes
  thrust::upper_bound(
    count, count + nsegments, labels.begin(), labels.end(), size_begin);
  thrust::adjacent_difference(size_begin, size_end, size_begin);

  // Access the chrominance using the sorted indices
  auto value_it = thrust::make_permutation_iterator(
    zip_it(di_chrominance.begin(), di_chrominance.begin() + npoints),
    indices.begin());
  auto total_it =
    zip_it(total_chrominance.begin(), total_chrominance.begin() + nsegments);
  auto discard_it = thrust::make_discard_iterator();
  // Reduce all segments to get their total chrominance
  thrust::reduce_by_key(labels.begin(),
                        labels.end(),
                        value_it,
                        discard_it,
                        total_it,
                        thrust::equal_to<int>(),
                        AddPair{});

  // Iterator to access the average chrominance of each segment
  auto average_chrominance = thrust::make_transform_iterator(
    zip_it(total_chrominance.begin(),
           total_chrominance.begin() + nsegments,
           size_begin),
    [] __device__(const thrust::tuple<real, real, int>& tc) {
      return thrust::make_tuple(tc.get<0>() / tc.get<2>(),
                                tc.get<1>() / tc.get<2>());
    });
  // Get matrix values as squared distance in chrominance space
  auto entry_it = thrust::make_transform_iterator(
    zip_it(thrust::make_permutation_iterator(
             average_chrominance, dio_segment_adjacency_keys.begin()),
           thrust::make_permutation_iterator(average_chrominance,
                                             dio_segment_adjacency.begin()),
           dio_segment_adjacency.begin()),
    ChominanceDistance2{});

  // Reduce by column to find the lowest distance, and hence nearest in
  // chrominance space to our segment, this is the segment we want to merge
  // with.
  thrust::reduce_by_key(dio_segment_adjacency_keys.begin(),
                        dio_segment_adjacency_keys.end(),
                        entry_it,
                        discard_it,
                        zip_it(discard_it, d_targets.begin()),
                        thrust::equal_to<int>(),
                        thrust::minimum<thrust::tuple<real, int>>());

  // We have converged if all segments have size greater than or equal to P
  auto has_converged = [old_labels = labels.begin(),
                        new_labels = dio_segment_labels.begin(),
                        npoints = npoints] {
    return thrust::equal(new_labels, new_labels + npoints, old_labels);
  };
  // Iterate over the target and current labels, with the current segment size,
  // using a transform functor to decide the final target to write
  auto target_it = thrust::make_transform_iterator(
    thrust::make_permutation_iterator(
      zip_it(d_targets.begin(), dio_segment_labels.begin(), size_begin),
      dio_segment_labels.begin()),
    TargetMap{P});
  // An iterator that provides a mapping from current to target labels
  auto map_it = zip_it(dio_segment_labels.begin(), target_it);
  // Loop until convergence
  while (!has_converged())
  {
    thrust::copy_n(dio_segment_labels.begin(), npoints, labels.begin());
    // Merge segments by replacing their labels with the target labels, if the
    // segment is small (size < P)
    thrust::transform(
      map_it,
      map_it + nsegments,
      // Targets targets
      thrust::make_permutation_iterator(d_targets.begin(), target_it),
      dio_segment_labels.begin(),
      MergeSegment{});
  }
}

}  // namespace ccl

SPLIT_DEVICE_NAMESPACE_END

