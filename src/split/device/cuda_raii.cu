#include "split/device/cuda_raii.cuh"

SPLIT_DEVICE_NAMESPACE_BEGIN

ScopedCuStream::ScopedCuStream()
{
  status = hipStreamCreate(&handle);
}

ScopedCuStream::~ScopedCuStream()
{
  join();
  status = hipStreamDestroy(handle);
}

ScopedCuStream::operator hipStream_t() const noexcept
{
  return handle;
}

void ScopedCuStream::join() noexcept
{
  status = hipStreamSynchronize(handle);
}

SPLIT_DEVICE_NAMESPACE_END
