#include "split/device/kmeans/kmeans.cuh"
#include "split/device/ccl/segment_adjacency.cuh"
#include "split/device/ccl/connected_components.cuh"
#include "split/device/ccl/compress_labels.cuh"
#include "split/device/ccl/segment_adjacency.cuh"
#include "split/device/ccl/merge_small_segments.cuh"
#include "split/device/ccl/merge_smooth_boundaries.cuh"
#include "split/device/color/conversion.cuh"
#include "split/device/color/beta_feature.cuh"
#include "split/device/detail/view_util.cuh"
#include "split/device/detail/unary_functional.cuh"
#include "split/device/detail/cycle_iterator.cuh"
#include "split/host/stbi/stbi_raii.hpp"
#include "split/device/detail/cu_raii.cuh"
#include "split/device/morph/erode.cuh"

#include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <thrust/iterator/transform_output_iterator.h>

#include <chrono>

#define real split::real

template <typename T>
void BAD_call_destructor(T& io_obj)
{
  io_obj.~T();
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
  switch (error)
  {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";

    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";

    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";

    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";

    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }

  return "<unknown>";
}

void sandbox()
{
  printf("\n\n\n\n\n SANDBOX\n\n");
  const int A = 0;
  const int B = 1;
  const int C = 2;
  const int D = 3;
  std::vector<int> labels = {
    A,A,A,A,A,A,A,
    A,A,A,A,B,B,B,
    A,A,A,A,B,B,B,
    A,A,B,B,B,B,B,
    C,C,B,B,B,B,B,
    C,C,C,C,C,D,D,
    C,C,C,C,C,D,D
  };
  const int npixels = 42;

  cusp::array2d<int, cusp::device_memory> d_labels(7, 7);
  thrust::copy_n(labels.begin(), npixels, d_labels.values.begin());

  split::device::morph::erode(d_labels, 2);

  cusp::print(d_labels);

}

template <typename T>
void strided_copy(const T* i_src,
                  T* i_dest,
                  int src_stride,
                  int dest_stride,
                  int n,
                  hipMemcpyKind i_kind)
{
  hipMemcpy2D(i_dest,
               sizeof(T) * dest_stride,
               i_src,
               sizeof(T) * src_stride,
               sizeof(T),
               n,
               i_kind);
}

void make_device_image(gsl::not_null<const real*> h_image,
                       cusp::array2d<real, cusp::device_memory>::view d_image)
{
  const int npixels = d_image.num_cols;
  const int nchannels = d_image.num_rows;
  for (int c = 0; c < nchannels; ++c)
  {
    auto d_image_channel = d_image.values.begin().base().get() + npixels * c;
    const auto h_image_channel = h_image.get() + c;
    strided_copy(h_image_channel,
                 d_image_channel,
                 nchannels,
                 1,
                 npixels,
                 hipMemcpyHostToDevice);
  }
}

void make_host_image(cusp::array2d<real, cusp::device_memory>::view d_image,
                     gsl::not_null<real*> h_image)
{
  const int npixels = d_image.num_cols;
  const int nchannels = d_image.num_rows;
  for (int c = 0; c < nchannels; ++c)
  {
    auto d_image_channel = d_image.values.begin().base().get() + npixels * c;
    const auto h_image_channel = h_image.get() + c;
    strided_copy(d_image_channel,
                 h_image_channel,
                 1,
                 nchannels,
                 npixels,
                 hipMemcpyDeviceToHost);
  }
}

struct TempMemory
{
  TempMemory(std::size_t N) : m_ptr(thrust::device_malloc(N))
  {
  }

  ~TempMemory()
  {
    thrust::device_free(m_ptr);
  }

  thrust::device_ptr<void> get() const
  {
    return m_ptr;
  }

private:
  thrust::device_ptr<void> m_ptr;
};

int main(int argc, char* argv[])
{
#if 0
  assert(argc == 2);
  auto h_image = split::host::stbi::loadf(argv[1], 3);
  printf("Loaded image with dim: %dx%dx%d\n",
         h_image.width(),
         h_image.height(),
         h_image.n_channels());

  cusp::array2d<real, cusp::device_memory> d_rgb_image(h_image.n_channels(),
                                                       h_image.n_pixels());
  cusp::array2d<real, cusp::device_memory> d_lab_image(h_image.n_channels(),
                                                       h_image.n_pixels());
  make_device_image(h_image.get(), d_rgb_image);

  // Convert the input linear RGB image into L*a*b color space
  split::device::color::convert_color_space(
    d_rgb_image, d_lab_image, split::device::color::rgb_to_lab());
  // Make a copy of the image luminance before writing the beta feature
  cusp::array1d<real, cusp::device_memory> d_luminance(h_image.n_pixels());
  thrust::copy(
    d_lab_image.row(0).begin(), d_lab_image.row(0).end(), d_luminance.begin());
  // Compute the beta feature from our lab image, and write it in place of L
  split::device::color::beta_feature(d_lab_image, d_lab_image.row(0));

  // Create initial means
  const int nclusters = 5;
  const int npixels = h_image.n_pixels();

  cusp::array2d<real, cusp::device_memory> d_centroids(h_image.n_channels(),
                                                       nclusters);
  split::device::kmeans::uniform_random_initialize(d_lab_image, d_centroids);

  cusp::array2d<int, cusp::device_memory> d_cluster_labels(h_image.height(),
                                                           h_image.width());
  cusp::array2d<int, cusp::device_memory> d_segment_labels(h_image.height(),
                                                           h_image.width());

  // Allocate temporary memory
  TempMemory d_temp(split::device::kmeans::cluster_workspace(
    npixels, nclusters, h_image.n_channels()));

  // K-means cluster the image
  split::device::kmeans::cluster(
    d_lab_image, d_centroids, d_cluster_labels.values, d_temp.get(), 100, 1e-3);

  // Obtain isolated segments from our initial clustering
  split::device::ccl::connected_components(
    d_cluster_labels, d_temp.get(), d_segment_labels.values, 100);
  // Compress the segment labels to produce a contiguous sequence
  int nsegments =
    split::device::ccl::compress_labels(d_segment_labels.values, d_temp.get());
  std::cout << "Segmented into " << nsegments << " connected components.\n";
  // Re-calculate the centroids using the segment labels
  cusp::array2d<real, cusp::device_memory> d_seg_centroids(h_image.n_channels(),
                                                           nsegments);

  // Re-calculate the centroids using the segment labels
  split::device::kmeans::calculate_centroids(
    d_cluster_labels.values, d_rgb_image, d_centroids, d_temp.get());
  //---------------------------------------------------------------------------
  auto d_chrominance = split::device::detail::make_const_array2d_view(
    cusp::make_array2d_view(2,
                            npixels,
                            npixels,
                            d_lab_image.values.subarray(npixels, npixels * 2),
                            cusp::row_major{}));

  for (int i = 0; i < 10 && (nsegments > 5000 || i < 1); ++i)
  {
    TempMemory d_temp(
      split::device::ccl::merge_small_segments_workspace(npixels, nsegments));
    std::cout << "Merging small clusters\n";
    split::device::ccl::merge_small_segments(
      d_chrominance, d_segment_labels, d_temp.get(), 10 * (i + 1));
    nsegments = split::device::ccl::compress_labels(d_segment_labels.values,
                                                    d_temp.get());
  }
  printf("Number of segments post merge: %d\n", nsegments);

  // Sanity check
  make_host_image(d_rgb_image, h_image.get());
  split::host::stbi::writef("assets/images/sanity.png", h_image);

  // Copy the segment means to their member pixels
  split::device::kmeans::propagate_centroids(
    d_cluster_labels.values, d_centroids, d_rgb_image);
  make_host_image(d_rgb_image, h_image.get());
  split::host::stbi::writef("assets/images/clusters.png", h_image);

  // Re-calculate the centroids using the segment labels
  split::device::kmeans::calculate_centroids(
    d_segment_labels.values, d_rgb_image, d_seg_centroids, d_temp.get());
  // Copy the segment means to their member pixels
  split::device::kmeans::propagate_centroids(
    d_segment_labels.values, d_seg_centroids, d_rgb_image);

  make_host_image(d_rgb_image, h_image.get());
  split::host::stbi::writef("assets/images/components.png", h_image);

  //------------------------------------------------------------------
#else

  sandbox();
#endif

  return 0;
}

