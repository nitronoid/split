#include "split/device/kmeans/kmeans.cuh"
#include "split/host/stbi/stbi_raii.hpp"

#include <cusp/print.h>
#include <cusp/array1d.h>
#include <cusp/array2d.h>

#include <chrono>

#define real split::real

template <typename T>
void strided_copy(const T* i_src,
                  T* i_dest,
                  int src_stride,
                  int dest_stride,
                  int n,
                  hipMemcpyKind i_kind)
{
  hipMemcpy2D(i_dest,
               sizeof(T) * dest_stride,
               i_src,
               sizeof(T) * src_stride,
               sizeof(T),
               n,
               i_kind);
}

void make_device_image(gsl::not_null<const real*> h_image,
                       cusp::array2d<real, cusp::device_memory>::view d_image)
{
  const int npixels = d_image.num_cols;
  auto d_image_r = d_image.values.begin().base().get() + npixels * 0;
  auto d_image_g = d_image.values.begin().base().get() + npixels * 1;
  auto d_image_b = d_image.values.begin().base().get() + npixels * 2;
  const auto h_image_r = h_image.get() + 0;
  const auto h_image_g = h_image.get() + 1;
  const auto h_image_b = h_image.get() + 2;
  strided_copy(h_image_r, d_image_r, 3, 1, npixels, hipMemcpyHostToDevice);
  strided_copy(h_image_g, d_image_g, 3, 1, npixels, hipMemcpyHostToDevice);
  strided_copy(h_image_b, d_image_b, 3, 1, npixels, hipMemcpyHostToDevice);
}

void make_host_image(cusp::array2d<real, cusp::device_memory>::view d_image,
                     gsl::not_null<real*> h_image)
{
  const int npixels = d_image.num_cols;
  auto d_image_r = d_image.values.begin().base().get() + npixels * 0;
  auto d_image_g = d_image.values.begin().base().get() + npixels * 1;
  auto d_image_b = d_image.values.begin().base().get() + npixels * 2;
  const auto h_image_r = h_image.get() + 0;
  const auto h_image_g = h_image.get() + 1;
  const auto h_image_b = h_image.get() + 2;
  strided_copy(d_image_r, h_image_r, 1, 3, npixels, hipMemcpyDeviceToHost);
  strided_copy(d_image_g, h_image_g, 1, 3, npixels, hipMemcpyDeviceToHost);
  strided_copy(d_image_b, h_image_b, 1, 3, npixels, hipMemcpyDeviceToHost);
}

int main()
{
  auto h_image = split::host::stbi::loadf("assets/images/rust.png", 3);
  printf("Loaded image with dim: %dx%dx%d\n",
         h_image.width(),
         h_image.height(),
         h_image.n_channels());

  cusp::array2d<real, cusp::device_memory> d_image(h_image.n_channels(),
                                                   h_image.n_pixels());
  make_device_image(h_image.get(), d_image);
  std::fill_n(h_image.get(), h_image.n_pixel_data(), 0.5f);

  // Create initial means
  const int nclusters = 3;
  cusp::array2d<real, cusp::device_memory, cusp::column_major> d_centroids(
    nclusters, h_image.n_channels());
  std::cout << "Generating centroids\n";
  split::device::kmeans::initialize_centroids(d_image, d_centroids);
  cusp::print(d_centroids);
  std::cout << "Done\n";

  cusp::array1d<int, cusp::device_memory> d_cluster_labels(h_image.n_pixels());

  // Allocate temporary memory
  thrust::device_vector<uint8_t> d_temp(h_image.n_pixels() * nclusters *
                                        sizeof(real));

  thrust::device_ptr<void> d_temp_ptr{static_cast<void*>(d_temp.data().get())};

  split::device::kmeans::cluster(
    d_image, d_centroids, d_cluster_labels, d_temp_ptr, 100, 5e-1);

  std::cout << "Finalizing cluster colors\n";
  split::device::kmeans::propagate_centroids(
    d_cluster_labels, d_centroids, d_image);
  std::cout << "Done\n";

  make_host_image(d_image, h_image.get());

  split::host::stbi::writef("assets/images/out.png", h_image);

  return 0;
}

